
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/Sigmoid.cu"
#else

#include "../common.h"

void THNN_(Sigmoid_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *output)
{
  THCUNN_assertSameGPU(state, 2, input, output);
  THCTensor_(resizeAs)(state, output, input);
  THC_pointwiseApply2(state, output, input, sigmoidupdateOutput_functor<real>());
}

void THNN_(Sigmoid_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           THCTensor *output)
{
  THCUNN_check_nElement(state, input, gradOutput);
  THCUNN_assertSameGPU(state, 3, output, gradOutput, gradInput);
  THCTensor_(resizeAs)(state, gradInput, output);
  THC_pointwiseApply3(state, gradInput, output, gradOutput, sigmoidupdateGradInput_functor<real>());
}

#endif
